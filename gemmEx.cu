#include "hip/hip_runtime.h"
#include <hipblas.h>

#include <string>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/library_types.h>
#include <hip/hip_fp16.h>

#include "args.h"
#include "cuda_err.h"

/*
 * Special thanks to the following codes and posts for ideas 
 * on how to run this benchmark:
 * https://github.com/hma02/hipblasHgemm-P100
 * https://docs.nvidia.com/deeplearning/performance/dl-performance-matrix-multiplication/index.html
 */

/*
 * Kernels
 */

//Copy kernel simplified for experiments used here
//from Elemental/src/hydrogen/blas/gpu/Copy.cu line 28
template<typename T, typename U>
__global__ void copy_2d(
    int m, int n,
    T const* __restrict__ src, int src_row_stride, int src_col_stride,
    U* __restrict__ dest, int dest_row_stride, int dest_col_stride)
{
	const int TILE_SIZE = 32;
	const int BLK_COLS = 8;
    __shared__ T** tile_shared[TILE_SIZE][TILE_SIZE+1];
    auto tile = reinterpret_cast<T(*)[TILE_SIZE+1]>(tile_shared);

    int const start_row = blockIdx.x * TILE_SIZE + threadIdx.x;
    int const start_col = blockIdx.y * TILE_SIZE + threadIdx.y;

    src += start_row*src_row_stride + start_col*src_col_stride;
    dest += start_row*dest_row_stride + start_col*dest_col_stride;
    if (start_row < m && start_col < n)
    {
        if (start_col + TILE_SIZE < n)
        {
            // Load the data
            #pragma unroll
            for (int ii = 0; ii < TILE_SIZE; ii += BLK_COLS)
                tile[threadIdx.y+ii][threadIdx.x] = src[ii*src_col_stride];

            // Store the data
            #pragma unroll
            for (int ii = 0; ii < TILE_SIZE; ii += BLK_COLS)
                dest[ii*dest_col_stride] = tile[threadIdx.y+ii][threadIdx.x];
        }
        else
        {
            // Load the data
            for (int ii = 0; ii < TILE_SIZE && start_col + ii < n; ii += BLK_COLS)
            {
                tile[threadIdx.y+ii][threadIdx.x] = src[ii*src_col_stride];
            }

            // Store the data
            for (int ii = 0; ii < TILE_SIZE && start_col + ii < n; ii += BLK_COLS)
            {
                dest[ii*dest_col_stride] = tile[threadIdx.y+ii][threadIdx.x];
            }
        }
    }
}

template<typename T>
__global__ void initA(int n, T* A)
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < n; i+= stride)
		A[i] = T(2.0);
}

template<typename T>
__global__ void initB(int n, T* B)
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < n; i+= stride)
		B[i] = T(3.0);
}

template<typename T>
__global__ void initC(int n, T* C)
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < n; i+= stride)
		C[i] = T(0.0);
}

template<typename T>
__global__ void initConsts(T* a, T* b)
{
	*a = T(1.0);
	*b = T(0.0);
}

/*
 * Experiment definitions
 */

float copy_experiment(Args);
float gemmex_experiment(Args);

/*
 * Main
 */

int main(int argc, char* argv[])
{
	Args a;
	if (!get_args(argc, argv, &a))
		return 1;

#if defined(DEBUG)
	std::cout << "m: " << a.m << " n: " << a.n << " k: " << a.k;
	if (a.is_copy)
		std::cout << " copy: True" << std::endl;
	else
		std::cout << " copy: False" << std::endl;
#endif

	float t = 0.0f;
	if (a.is_copy)
	{
		for (int i = 0; i < a.n_runs; i++)
			t += copy_experiment(a);

		std::cout << "Copy + SGEMM Ave Elapsed Time: " 
			<< t/a.n_runs << "s" << std::endl;
	}
	else
	{
		for (int i = 0; i < a.n_runs; i++)
			t += gemmex_experiment(a);
		std::cout << "GemmEX Ave Elapsed Time: " 
			<< t/a.n_runs << "s" << std::endl;
	}

	return 0;
}

float copy_experiment(Args args)
{
	__half *A_orig;
	float *A;
	__half *B_orig;
	float *B;
	float *C;
	float *alpha;
	float *beta;

	//Initializations
	int a_size = args.m * args.k;
	int b_size = args.k * args.n;
	int c_size = args.m * args.n;
	checkCuda(hipMallocManaged(&A_orig, a_size*sizeof(__half)));
	checkCuda(hipMallocManaged(&A, a_size*sizeof(float)));
	checkCuda(hipMallocManaged(&B_orig, b_size*sizeof(__half)));
	checkCuda(hipMallocManaged(&B, b_size*sizeof(float)));
	checkCuda(hipMallocManaged(&C, c_size*sizeof(float)));
	checkCuda(hipMallocManaged(&alpha, sizeof(float)));
	checkCuda(hipMallocManaged(&beta, sizeof(float)));

	initA<<<1, 256>>>(a_size, A_orig);
	initB<<<1, 256>>>(b_size, B_orig);
	initC<<<1, 256>>>(c_size, C);
	initConsts<<<1, 1>>>(alpha, beta);

	//Experiment
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasStatus_t stat;
	hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
	copy_2d<<<1, 256>>>(args.m, args.k, 
		A_orig, 1, 256, //src
		A, 1, 256); //dest
	copy_2d<<<1, 256>>>(args.m, args.k, 
		B_orig, 1, 256, //src
		B, 1, 256); //dest
	stat = hipblasSgemm(handle,
		HIPBLAS_OP_N, HIPBLAS_OP_N,
		args.m, args.n, args.k,
		alpha,
		A, args.m,
		B, args.k,
		beta,
		C, args.n);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		std::cerr << "cublas gemm Failed...." << std::endl;
		std::cerr << checkCublas(stat) << std::endl;
		exit(1);
	}
	checkCuda(hipGetLastError());
	float elapsed;
	hipEventElapsedTime(&elapsed, start, stop);
	elapsed /= 1000.0f;

	//Freedom
	hipFree(A_orig);
	hipFree(A);
	hipFree(B);
	hipFree(B_orig);
	hipFree(C);
	hipFree(alpha);
	hipFree(beta);

	return elapsed;
}

float gemmex_experiment(Args args)
{
	__half *A;
	__half *B;
	float *C;
	float *alpha;
	float *beta;

	//Initializations
	int a_size = args.m * args.k;
	int b_size = args.k * args.n;
	int c_size = args.m * args.n;
	checkCuda(hipMallocManaged(&A, a_size*sizeof(__half)));
	checkCuda(hipMallocManaged(&B, b_size*sizeof(__half)));
	checkCuda(hipMallocManaged(&C, c_size*sizeof(float)));
	checkCuda(hipMallocManaged(&alpha, sizeof(float)));
	checkCuda(hipMallocManaged(&beta, sizeof(float)));

	initA<<<1, 256>>>(a_size, A);
	initB<<<1, 256>>>(b_size, B);
	initC<<<1, 256>>>(c_size, C);
	initConsts<<<1, 1>>>(alpha, beta);

	//Experiment
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasStatus_t stat;
	hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
	stat = hipblasGemmEx(handle,
		HIPBLAS_OP_N, HIPBLAS_OP_N,
		args.m, args.n, args.k,
		alpha,
		A, HIP_R_16F, args.m,
		B, HIP_R_16F, args.k,
		beta,
		C, HIP_R_32F, args.n,
		HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		std::cerr << "hipblasGemmEx Failed...." << std::endl;
		std::cerr << "cublas Err no: " << stat << std::endl;
		std::cerr << checkCublas(stat) << std::endl;
		exit(1);
	}
	checkCuda(hipGetLastError());
	float elapsed;
	hipEventElapsedTime(&elapsed, start, stop);
	elapsed /= 1000.0f;

	//Freedom
	hipFree(A);
	hipFree(B);
	hipFree(C);
	hipFree(alpha);
	hipFree(beta);

	return elapsed;
}